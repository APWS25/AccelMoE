#include "hip/hip_runtime.h"
#include "model.h"


/* [Tensor Structure] */
/* Tensor
 * @brief - A multi-dimensional matrix containing elements of a single data
 type.
 * @member - buf  : Data buffer containing elements
 * @member - shape: Shape of tensor from outermost dimension to innermost
 dimension e.g., {{1.0, -0.5, 2.3}, {4.3, 5.6, -7.8}} => shape = {2, 3}
 */

 // 생성자: 기본 버전
Tensor::Tensor(const vector<size_t> &shape_) {
  ndim = shape_.size();
  for (size_t i = 0; i < ndim; i++) { shape[i] = shape_[i]; }
  size_t N_ = num_elem();
  // CPU
  CHECK_CUDA(hipHostMalloc((void**)&buf, sizeof(float) * N_)); // pinned
  // GPU
  CHECK_CUDA(hipMalloc((void**)&gbuf, sizeof(float) * N_)); 
  CHECK_CUDA(hipMemset(gbuf, 0, sizeof(float) * N_));
}

// 생성자: 기존 버퍼 사용
Tensor::Tensor(const vector<size_t> &shape_, float *buf_) {
  ndim = shape_.size();
  for (size_t i = 0; i < ndim; i++) { shape[i] = shape_[i]; }
  size_t N_ = num_elem();
  // CPU
  CHECK_CUDA(hipHostMalloc((void**)&buf, sizeof(float) * N_)); // pinned
  memcpy(buf, buf_, N_ * sizeof(float));
  // GPU
  CHECK_CUDA(hipMalloc((void**)&gbuf, sizeof(float) * N_)); 
  CHECK_CUDA(hipMemcpy(gbuf, buf_, N_ * sizeof(float), hipMemcpyHostToDevice));
}

void Tensor::compare() {
  size_t N_ = num_elem();
  float* tmp = new float[N_];
  CHECK_CUDA(hipMemcpy(tmp, gbuf, N_ * sizeof(float), hipMemcpyDeviceToHost));
  
  for (size_t i = 0; i < N_; i++) {
      if (fabs(buf[i] - tmp[i]) > 1e-5) {
          printf("Mismatch at index %zu: CPU: %f GPU: %f\n", i, buf[i], tmp[i]);
      }
  }
  delete[] tmp;
}

// GPU에서 CPU로 데이터 복사
void Tensor::toCPU() {
  size_t N_ = num_elem();
  CHECK_CUDA(hipMemcpy(buf, gbuf, N_ * sizeof(float), hipMemcpyDeviceToHost));
}

// CPU에서 GPU로 데이터 복사
void Tensor::toGPU() {
  size_t N_ = num_elem();
  CHECK_CUDA(hipMemcpy(gbuf, buf, N_ * sizeof(float), hipMemcpyHostToDevice));
}

// Tensor 소멸자
Tensor::~Tensor() {
  if (buf != nullptr) CHECK_CUDA(hipHostFree(buf));
  if (gbuf != nullptr) CHECK_CUDA(hipFree(gbuf));
}

// 텐서의 총 원소 개수를 반환하는 함수
size_t Tensor::num_elem() {
  size_t size = 1;
  for (size_t i = 0; i < ndim; i++) { size *= shape[i]; }
  return size;
}